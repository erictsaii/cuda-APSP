#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEV_NO 0
#define B 64
#define half_B 32
 
hipDeviceProp_t prop;
int cpu_cnt;

int* D;
int V;
int E;

void input(const char* filename) {
    // open file
    FILE* file = fopen(filename, "rb");
	fread(&V, sizeof(int), 1, file);
	fread(&E, sizeof(int), 1, file);

    // initialize matrix
    D = (int *)malloc(V * V * sizeof(int));
    for (int i = 0; i < V; ++i){
        int i_V = i * V;
        for (int j = 0; j < V; ++j){
            if (i == j) D[i_V + j] = 0;
            else D[i_V + j] = 1073741823;
        }
    }
    int tmp[300];
    if (E >= 100){
        int j = 0;
        for (; j < E; j += 100) {
            fread(tmp, sizeof(int), 300, file);
            for (int i = 0; i < 300; i += 3){
                D[tmp[i] * V + tmp[i+1]] = tmp[i+2];
            }
	    }
        for (int i = j - 100; i < E; ++i) {
            fread(tmp, sizeof(int), 3, file);
            D[tmp[0] * V + tmp[1]] = tmp[2];
	    }
    }
    else{
        for (int i = 0; i < E; ++i) {
            fread(tmp, sizeof(int), 3, file);
            D[tmp[0] * V + tmp[1]] = tmp[2];
	    }
    }
	fclose(file);
}


void output(const char* filename) {
    FILE* file = fopen(filename, "w");
    fwrite(D, sizeof(int), V * V, file);
	fclose(file);
}

__global__ void phase_1(int *d_D, int round, int V) {
    __shared__ int share_D[B * B];
    // for share
    const int s_x = threadIdx.x;
    const int s_y = threadIdx.y;
    // for global
    const int g_x = round * B + threadIdx.x; 
    const int g_y = round * B + threadIdx.y; 
    // block size is 64*64 but we only have 32*32 threads 
    // 1 thread deals with 4 kinds of D[i, j]
    share_D[s_y * B + s_x] = d_D[g_y * V + g_x];
    share_D[s_y * B + (s_x + half_B)] = d_D[g_y * V + (g_x + half_B)];
    share_D[(s_y + half_B) * B + s_x] = d_D[(g_y + half_B) * V + g_x];
    share_D[(s_y + half_B) * B + (s_x + half_B)] = d_D[(g_y + half_B) * V + (g_x + half_B)];

    __syncthreads();

    #pragma unroll 32 //necessary?
	for (int k = 0; k < B; ++k) {
		share_D[s_y * B + s_x] = min(share_D[s_y * B + s_x], share_D[s_y * B + k] + share_D[k * B + s_x]);
        share_D[s_y * B + (s_x + half_B)] = min(share_D[s_y * B + (s_x + half_B)], share_D[s_y * B + k] + share_D[k * B + (s_x + half_B)]);
		share_D[(s_y + half_B) * B + s_x] = min(share_D[(s_y + half_B) * B + s_x], share_D[(s_y + half_B) * B + k] + share_D[k * B + s_x]);
		share_D[(s_y + half_B) * B + (s_x + half_B)] = min(share_D[(s_y + half_B) * B + (s_x + half_B)], share_D[(s_y + half_B) * B + k] + share_D[k * B + (s_x + half_B)]);
		__syncthreads();
	}

    // load back to global
	d_D[g_y * V + g_x] = share_D[s_y * B + s_x];
    d_D[g_y * V + (g_x + half_B)] = share_D[s_y * B + (s_x + half_B)]; 
    d_D[(g_y + half_B) * V + g_x] = share_D[(s_y + half_B) * B + s_x];
    d_D[(g_y + half_B) * V + (g_x + half_B)] = share_D[(s_y + half_B) * B + (s_x + half_B)];
}

__global__ void phase_2(int *d_D, int round, int V) {
    if (blockIdx.x == round) return;
    // init share memory
    __shared__ int pivot_D[B * B];
    __shared__ int row_D[B * B];
    __shared__ int col_D[B * B];
    // load pivot_D
    const int s_x = threadIdx.x;
    const int s_y = threadIdx.y;
    int g_x = round * B + threadIdx.x; 
    int g_y = round * B + threadIdx.y;

    pivot_D[s_y * B + s_x] = d_D[g_y * V + g_x];
    pivot_D[s_y * B + (s_x + half_B)] = d_D[g_y * V + (g_x + half_B)];
    pivot_D[(s_y + half_B) * B + s_x] = d_D[(g_y + half_B) * V + g_x];
    pivot_D[(s_y + half_B) * B + (s_x + half_B)] = d_D[(g_y + half_B) * V + (g_x + half_B)];
    
    // load row_D
    g_x = blockIdx.x * B + threadIdx.x; 
    g_y = round * B + threadIdx.y;

    row_D[s_y * B + s_x] = d_D[g_y * V + g_x];
    row_D[s_y * B + (s_x + half_B)] = d_D[g_y * V + (g_x + half_B)];
    row_D[(s_y + half_B) * B + s_x] = d_D[(g_y + half_B) * V + g_x];
    row_D[(s_y + half_B) * B + (s_x + half_B)] = d_D[(g_y + half_B) * V + (g_x + half_B)];

    // load col_D
    g_x = round * B + threadIdx.x; 
    g_y = blockIdx.x * B + threadIdx.y;

    col_D[s_y * B + s_x] = d_D[g_y * V + g_x];
    col_D[s_y * B + (s_x + half_B)] = d_D[g_y * V + (g_x + half_B)];
    col_D[(s_y + half_B) * B + s_x] = d_D[(g_y + half_B) * V + g_x];
    col_D[(s_y + half_B) * B + (s_x + half_B)] = d_D[(g_y + half_B) * V + (g_x + half_B)];

    __syncthreads();

    // calculate
    #pragma unroll 32
    for (int k = 0; k < B; ++k){
        // row
        row_D[s_y * B + s_x] = min(row_D[s_y * B + s_x], pivot_D[s_y * B + k] + row_D[k * B + s_x]);
        row_D[s_y * B + (s_x + half_B)] = min(row_D[s_y * B + (s_x + half_B)], pivot_D[s_y * B + k] + row_D[k * B + (s_x + half_B)]);
		row_D[(s_y + half_B) * B + s_x] = min(row_D[(s_y + half_B) * B + s_x], pivot_D[(s_y + half_B) * B + k] + row_D[k * B + s_x]);
		row_D[(s_y + half_B) * B + (s_x + half_B)] = min(row_D[(s_y + half_B) * B + (s_x + half_B)], pivot_D[(s_y + half_B) * B + k] + row_D[k * B + (s_x + half_B)]);
        // col
        col_D[s_y * B + s_x] = min(col_D[s_y * B + s_x], pivot_D[s_y * B + k] + col_D[k * B + s_x]);
        col_D[s_y * B + (s_x + half_B)] = min(col_D[s_y * B + (s_x + half_B)], pivot_D[s_y * B + k] + col_D[k * B + (s_x + half_B)]);
		col_D[(s_y + half_B) * B + s_x] = min(col_D[(s_y + half_B) * B + s_x], pivot_D[(s_y + half_B) * B + k] + col_D[k * B + s_x]);
		col_D[(s_y + half_B) * B + (s_x + half_B)] = min(col_D[(s_y + half_B) * B + (s_x + half_B)], pivot_D[(s_y + half_B) * B + k] + col_D[k * B + (s_x + half_B)]);
        // no __syncthreads()?
    }
    // load col back to global
    d_D[g_y * V + g_x] = col_D[s_y * B + s_x];
    d_D[g_y * V + (g_x + half_B)] = col_D[s_y * B + (s_x + half_B)];
    d_D[(g_y + half_B) * V + g_x] = col_D[(s_y + half_B) * B + s_x];
    d_D[(g_y + half_B) * V + (g_x + half_B)] = col_D[(s_y + half_B) * B + (s_x + half_B)];

    // load row back to global
    g_x = blockIdx.x * B + threadIdx.x; 
    g_y = round * B + threadIdx.y;

    d_D[g_y * V + g_x] = row_D[s_y * B + s_x];
    d_D[g_y * V + (g_x + half_B)] = row_D[s_y * B + (s_x + half_B)];
    d_D[(g_y + half_B) * V + g_x] = row_D[(s_y + half_B) * B + s_x];
    d_D[(g_y + half_B) * V + (g_x + half_B)] = row_D[(s_y + half_B) * B + (s_x + half_B)];
}

int main(int argc, char** argv) {
    // get input
    input(argv[1]);

    // cudaGetDeviceProperties(&prop, DEV_NO);
    // printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);

    int* d_D;
    // pin host D, maybe cudaHostRegisterReadOnly?
    hipHostRegister(D, V * V * sizeof(int), hipHostRegisterDefault);
    hipMalloc(&d_D, V * V * sizeof(int));
    hipMemcpy(d_D, D, V * V * sizeof(int), hipMemcpyHostToDevice);

    // block
    // B 32 or 64?
    int rounds = V / B;
    dim3 num_threads(32, 32); // maybe 64*16 is faster 
    dim3 phase_2_blocks(rounds, 1);
    
    for (int i = 0; i < rounds; ++i) {
        phase_1<<<1, num_threads>>>(d_D, i, V);
        phase_2<<<phase_2_blocks, num_threads>>>(d_D, i, V);
    }

    // output
    output(argv[2]);
}