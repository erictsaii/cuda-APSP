
#include <hip/hip_runtime.h>
#include <immintrin.h>
#include <omp.h>
#include <pthread.h>

#include <cassert>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <queue>
#include <thread>
#include <utility>
#include <vector>

#ifdef DEBUG
#define DEBUG_PRINT(fmt, args...) fprintf(stderr, fmt, ##args);
#define DEBUG_MSG(str) std::cout << str << "\n";
#define CUDA_EXE(F)                                                \
    {                                                              \
        cudaError_t err = F;                                       \
        if ((err != cudaSuccess)) {                                \
            printf("Error %s at %s:%d\n", cudaGetErrorString(err), \
                   __FILE__, __LINE__);                            \
            exit(-1);                                              \
        }                                                          \
    }
#define CUDA_CHECK()                                                                    \
    {                                                                                   \
        cudaError_t err = cudaGetLastError();                                           \
        if ((err != cudaSuccess)) {                                                     \
            printf("Error %s at %s:%d\n", cudaGetErrorString(err), __FILE__, __LINE__); \
            exit(-1);                                                                   \
        }                                                                               \
    }
#else
#define DEBUG_PRINT(fmt, args...)
#define DEBUG_MSG(str)
#define CUDA_EXE(F) F;
#define CUDA_CHECK()
#endif  // DEBUG

#ifdef TIMING
#include <ctime>
#define TIMING_START(arg)          \
    struct timespec __start_##arg; \
    clock_gettime(CLOCK_MONOTONIC, &__start_##arg);
#define TIMING_END(arg)                                                                       \
    {                                                                                         \
        struct timespec __temp_##arg, __end_##arg;                                            \
        double __duration_##arg;                                                              \
        clock_gettime(CLOCK_MONOTONIC, &__end_##arg);                                         \
        if ((__end_##arg.tv_nsec - __start_##arg.tv_nsec) < 0) {                              \
            __temp_##arg.tv_sec = __end_##arg.tv_sec - __start_##arg.tv_sec - 1;              \
            __temp_##arg.tv_nsec = 1000000000 + __end_##arg.tv_nsec - __start_##arg.tv_nsec;  \
        } else {                                                                              \
            __temp_##arg.tv_sec = __end_##arg.tv_sec - __start_##arg.tv_sec;                  \
            __temp_##arg.tv_nsec = __end_##arg.tv_nsec - __start_##arg.tv_nsec;               \
        }                                                                                     \
        __duration_##arg = __temp_##arg.tv_sec + (double)__temp_##arg.tv_nsec / 1000000000.0; \
        printf("%s took %lfs.\n", #arg, __duration_##arg);                                    \
    }
#else
#define TIMING_START(arg)
#define TIMING_END(arg)
#endif  // TIMING

#define TILE 26
#define block_size 78
#define div_block 3
const int INF = ((1 << 30) - 1);

__device__ int blk_idx(int r, int c, int blk_pitch, int nblocks);

__global__ void proc_1_glob(int *blk_dist, int k, int blk_pitch, int nblocks);
__global__ void proc_2_glob(int *blk_dist, int s, int k, int blk_pitch, int nblocks);
__global__ void proc_3_glob(int *blk_dist, int s_i, int s_j, int k, int blk_pitch, int nblocks);

__global__ void init_dist(int *blk_dist, int blk_pitch, int nblocks);
__global__ void build_dist(int *edge, int E, int *blk_dist, int blk_pitch, int nblocks);
__global__ void copy_dist(int *blk_dist, int blk_pitch, int *dist, int pitch, int nblocks);

__global__ void proc_1_blk_glob(int *blk_dist, int k, int pitch);
__global__ void proc_2_blk_glob(int *blk_dist, int s, int k, int pitch);
__global__ void proc_3_blk_glob(int *blk_dist, int s_i, int s_j, int k, int pitch);

__global__ void init_blk_dist(int *blk_dist, int pitch);
__global__ void build_blk_dist(int *edge, int E, int *blk_dist, int pitch);

int main(int argc, char **argv) {
    auto compute_start = std::chrono::steady_clock::now();
    assert(argc == 3);

    char *input_filename = argv[1];
    char *output_filename = argv[2];
    FILE *input_file;
    FILE *output_file;
    int ncpus = omp_get_max_threads();
    int device_cnt;
    int V, E;
    int *edge;
    int *dist;
    int VP;
    int nblocks;
    hipDeviceProp_t deviceProp;

    hipGetDeviceCount(&device_cnt);
    hipSetDevice(0);
    hipGetDeviceProperties(&deviceProp, 0);

    TIMING_START(hw3_2);

    /* input */
    TIMING_START(input);
    input_file = fopen(input_filename, "rb");
    assert(input_file);
    fread(&V, sizeof(int), 1, input_file);
    fread(&E, sizeof(int), 1, input_file);
    edge = (int *)malloc(sizeof(int) * 3 * E);
    fread(edge, sizeof(int), 3 * E, input_file);
    dist = (int *)malloc(sizeof(int) * V * V);
    fclose(input_file);
    DEBUG_PRINT("vertices: %d\nedges: %d\n", V, E);
    TIMING_END(input);

    nblocks = (int)ceilf(float(V) / block_size);
    VP = nblocks * block_size;

    /* calculate */
    if ((size_t)VP * VP * 2 + 2 * 3 * E + V * V <= deviceProp.totalGlobalMem / 4) {
        DEBUG_PRINT("Flatten\n");
        int *edge_dev;
        int *dist_dev;
        int *blk_dist_dev;
        size_t blk_pitch;

        TIMING_START(calculate);

        hipHostRegister(edge, sizeof(int) * 3 * E, hipHostRegisterReadOnly);
        hipMalloc(&edge_dev, sizeof(int) * 3 * E);
        hipHostRegister(dist, sizeof(int) * V * V, hipHostRegisterDefault);
        hipMalloc(&blk_dist_dev, sizeof(int) * block_size * block_size * nblocks * nblocks);
        blk_pitch = block_size * block_size;

        hipMemcpy(edge_dev, edge, sizeof(int) * 3 * E, hipMemcpyDefault);

        init_dist<<<dim3(VP / TILE, VP / TILE), dim3(TILE, TILE)>>>(blk_dist_dev, blk_pitch, nblocks);
        build_dist<<<(int)ceilf((float)E / (TILE * TILE)), TILE * TILE>>>(edge_dev, E, blk_dist_dev, blk_pitch, nblocks);
        hipFree(edge_dev);

        dim3 blk(TILE, TILE);
        for (int k = 0, nk = nblocks - 1; k < nblocks; k++, nk--) {
            /* Phase 1 */
            proc_1_glob<<<1, blk>>>(blk_dist_dev, k, blk_pitch, nblocks);
            /* Phase 2 */
            proc_2_glob<<<dim3(nblocks - 1, 2), blk>>>(blk_dist_dev, 0, k, blk_pitch, nblocks);
            /* Phase 3 */
            proc_3_glob<<<dim3(nblocks - 1, nblocks - 1), blk>>>(blk_dist_dev, 0, 0, k, blk_pitch, nblocks);
        }

        hipMalloc(&dist_dev, sizeof(int) * VP * VP);
        copy_dist<<<dim3(VP / TILE, VP / TILE), dim3(TILE, TILE)>>>(blk_dist_dev, blk_pitch, dist_dev, VP, nblocks);
        hipMemcpy2D(dist, sizeof(int) * V, dist_dev, sizeof(int) * VP, sizeof(int) * V, V, hipMemcpyDefault);

        hipDeviceSynchronize();

        hipFree(blk_dist_dev);
        hipFree(dist_dev);

        TIMING_END(calculate);
    } else {
        int *edge_dev;
        int *dist_dev;

        TIMING_START(calculate);

        hipHostRegister(edge, sizeof(int) * 3 * E, hipHostRegisterReadOnly);
        hipMalloc(&edge_dev, sizeof(int) * 3 * E);
        hipHostRegister(dist, sizeof(int) * V * V, hipHostRegisterDefault);
        hipMalloc(&dist_dev, sizeof(int) * VP * VP);

        hipMemcpy(edge_dev, edge, sizeof(int) * 3 * E, hipMemcpyDefault);

        init_blk_dist<<<dim3(VP / TILE, VP / TILE), dim3(TILE, TILE)>>>(dist_dev, VP);
        build_blk_dist<<<(int)ceilf((float)E / (TILE * TILE)), TILE * TILE>>>(edge_dev, E, dist_dev, VP);
        hipFree(edge_dev);

        dim3 blk(TILE, TILE);
        for (int k = 0, nk = nblocks - 1; k < nblocks; k++, nk--) {
            /* Phase 1 */
            proc_1_blk_glob<<<1, blk>>>(dist_dev, k, VP);
            /* Phase 2 */
            proc_2_blk_glob<<<dim3(nblocks - 1, 2), blk>>>(dist_dev, 0, k, VP);
            /* Phase 3 */
            proc_3_blk_glob<<<dim3(nblocks - 1, nblocks - 1), blk>>>(dist_dev, 0, 0, k, VP);
        }

        hipMemcpy2D(dist, sizeof(int) * V, dist_dev, sizeof(int) * VP, sizeof(int) * V, V, hipMemcpyDefault);

        hipDeviceSynchronize();
        hipFree(dist_dev);

        TIMING_END(calculate);
    }
    /* output */
    TIMING_START(output);
    output_file = fopen(output_filename, "w");
    assert(output_file);
    fwrite(dist, 1, sizeof(int) * V * V, output_file);
    fclose(output_file);
    TIMING_END(output);
    TIMING_END(hw3_2);

    free(edge);
    free(dist);
    auto compute_end = std::chrono::steady_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(compute_end - compute_start);
    printf("time %lld ms\n", duration.count());
    return 0;
}

__device__ int blk_idx(int r, int c, int blk_pitch, int nblocks) {
    return ((r / block_size) * nblocks + (c / block_size)) * blk_pitch + (r % block_size) * block_size + (c % block_size);
}

#define _ref(i, j, r, c) blk_dist[(i * nblocks + j) * blk_pitch + (r)*block_size + c]
__global__ void proc_1_glob(int *blk_dist, int k, int blk_pitch, int nblocks) {
    __shared__ int k_k_sm[block_size][block_size];

    int r = threadIdx.y;
    int c = threadIdx.x;

#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_k_sm[r + rr * TILE][c + cc * TILE] = _ref(k, k, r + rr * TILE, c + cc * TILE);
        }
    }
    __syncthreads();

#pragma unroll
    for (int b = 0; b < block_size; b++) {
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                k_k_sm[r + rr * TILE][c + cc * TILE] = min(k_k_sm[r + rr * TILE][c + cc * TILE], k_k_sm[r + rr * TILE][b] + k_k_sm[b][c + cc * TILE]);
            }
        }
        __syncthreads();
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            _ref(k, k, r + rr * TILE, c + cc * TILE) = k_k_sm[r + rr * TILE][c + cc * TILE];
        }
    }
}
__global__ void proc_2_glob(int *blk_dist, int s, int k, int blk_pitch, int nblocks) {
    __shared__ int k_k_sm[block_size][block_size];
    __shared__ int sm[block_size][block_size];

    int i = s + blockIdx.x;
    int r = threadIdx.y;
    int c = threadIdx.x;

    if (i >= k)
        i++;

#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_k_sm[r + rr * TILE][c + cc * TILE] = _ref(k, k, r + rr * TILE, c + cc * TILE);
        }
    }
    if (blockIdx.y == 0) {
        /* rows */
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                sm[r + rr * TILE][c + cc * TILE] = _ref(i, k, r + rr * TILE, c + cc * TILE);
            }
        }
        __syncthreads();

#pragma unroll
        for (int b = 0; b < block_size; b++) {
#pragma unroll
            for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
                for (int cc = 0; cc < div_block; cc++) {
                    sm[r + rr * TILE][c + cc * TILE] = min(sm[r + rr * TILE][c + cc * TILE], sm[r + rr * TILE][b] + k_k_sm[b][c + cc * TILE]);
                }
            }
            __syncthreads();
        }
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                _ref(i, k, r + rr * TILE, c + cc * TILE) = sm[r + rr * TILE][c + cc * TILE];
            }
        }
    } else {
        /* cols */
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                sm[r + rr * TILE][c + cc * TILE] = _ref(k, i, r + rr * TILE, c + cc * TILE);
            }
        }
        __syncthreads();

#pragma unroll
        for (int b = 0; b < block_size; b++) {
#pragma unroll
            for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
                for (int cc = 0; cc < div_block; cc++) {
                    sm[r + rr * TILE][c + cc * TILE] = min(sm[r + rr * TILE][c + cc * TILE], k_k_sm[r + rr * TILE][b] + sm[b][c + cc * TILE]);
                }
            }
            __syncthreads();
        }
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                _ref(k, i, r + rr * TILE, c + cc * TILE) = sm[r + rr * TILE][c + cc * TILE];
            }
        }
    }
}
__global__ void proc_3_glob(int *blk_dist, int s_i, int s_j, int k, int blk_pitch, int nblocks) {
    __shared__ int i_k_sm[block_size][block_size];
    __shared__ int k_j_sm[block_size][block_size];

    int i = s_i + blockIdx.y;
    int j = s_j + blockIdx.x;
    int r = threadIdx.y;
    int c = threadIdx.x;
    int loc[div_block][div_block];

    if (i >= k)
        i++;
    if (j >= k)
        j++;

#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            i_k_sm[r + rr * TILE][c + cc * TILE] = _ref(i, k, r + rr * TILE, c + cc * TILE);
        }
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_j_sm[r + rr * TILE][c + cc * TILE] = _ref(k, j, r + rr * TILE, c + cc * TILE);
        }
    }
    __syncthreads();
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            loc[rr][cc] = _ref(i, j, r + rr * TILE, c + cc * TILE);
        }
    }

#pragma unroll
    for (int b = 0; b < block_size; b++) {
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                loc[rr][cc] = min(loc[rr][cc], i_k_sm[r + rr * TILE][b] + k_j_sm[b][c + cc * TILE]);
            }
        }
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            _ref(i, j, r + rr * TILE, c + cc * TILE) = loc[rr][cc];
        }
    }
}
__global__ void init_dist(int *blk_dist, int blk_pitch, int nblocks) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    blk_dist[blk_idx(r, c, blk_pitch, nblocks)] = (r != c) * INF;
}
__global__ void build_dist(int *edge, int E, int *blk_dist, int blk_pitch, int nblocks) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < E) {
        int src = *(edge + idx * 3);
        int dst = *(edge + idx * 3 + 1);
        int w = *(edge + idx * 3 + 2);
        blk_dist[blk_idx(src, dst, blk_pitch, nblocks)] = w;
    }
}
__global__ void copy_dist(int *blk_dist, int blk_pitch, int *dist, int pitch, int nblocks) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    dist[r * pitch + c] = blk_dist[blk_idx(r, c, blk_pitch, nblocks)];
}

#define _ref_blk(i, j, r, c) blk_dist[i * block_size * pitch + j * block_size + (r)*pitch + c]
__global__ void proc_1_blk_glob(int *blk_dist, int k, int pitch) {
    __shared__ int k_k_sm[block_size][block_size];

    int r = threadIdx.y;
    int c = threadIdx.x;

#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_k_sm[r + rr * TILE][c + cc * TILE] = _ref_blk(k, k, r + rr * TILE, c + cc * TILE);
        }
    }
    __syncthreads();

#pragma unroll
    for (int b = 0; b < block_size; b++) {
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                k_k_sm[r + rr * TILE][c + cc * TILE] = min(k_k_sm[r + rr * TILE][c + cc * TILE], k_k_sm[r + rr * TILE][b] + k_k_sm[b][c + cc * TILE]);
            }
        }
        __syncthreads();
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            _ref_blk(k, k, r + rr * TILE, c + cc * TILE) = k_k_sm[r + rr * TILE][c + cc * TILE];
        }
    }
}
__global__ void proc_2_blk_glob(int *blk_dist, int s, int k, int pitch) {
    __shared__ int k_k_sm[block_size][block_size];
    __shared__ int sm[block_size][block_size];

    int i = s + blockIdx.x;
    int r = threadIdx.y;
    int c = threadIdx.x;

    if (i >= k)
        i++;

#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_k_sm[r + rr * TILE][c + cc * TILE] = _ref_blk(k, k, r + rr * TILE, c + cc * TILE);
        }
    }
    if (blockIdx.y == 0) {
        /* rows */
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                sm[r + rr * TILE][c + cc * TILE] = _ref_blk(i, k, r + rr * TILE, c + cc * TILE);
            }
        }
        __syncthreads();

#pragma unroll
        for (int b = 0; b < block_size; b++) {
#pragma unroll
            for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
                for (int cc = 0; cc < div_block; cc++) {
                    sm[r + rr * TILE][c + cc * TILE] = min(sm[r + rr * TILE][c + cc * TILE], sm[r + rr * TILE][b] + k_k_sm[b][c + cc * TILE]);
                }
            }
            __syncthreads();
        }
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                _ref_blk(i, k, r + rr * TILE, c + cc * TILE) = sm[r + rr * TILE][c + cc * TILE];
            }
        }
    } else {
        /* cols */
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                sm[r + rr * TILE][c + cc * TILE] = _ref_blk(k, i, r + rr * TILE, c + cc * TILE);
            }
        }
        __syncthreads();

#pragma unroll
        for (int b = 0; b < block_size; b++) {
#pragma unroll
            for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
                for (int cc = 0; cc < div_block; cc++) {
                    sm[r + rr * TILE][c + cc * TILE] = min(sm[r + rr * TILE][c + cc * TILE], k_k_sm[r + rr * TILE][b] + sm[b][c + cc * TILE]);
                }
            }
            __syncthreads();
        }
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                _ref_blk(k, i, r + rr * TILE, c + cc * TILE) = sm[r + rr * TILE][c + cc * TILE];
            }
        }
    }
}
__global__ void proc_3_blk_glob(int *blk_dist, int s_i, int s_j, int k, int pitch) {
    __shared__ int i_k_sm[block_size][block_size];
    __shared__ int k_j_sm[block_size][block_size];

    int i = s_i + blockIdx.y;
    int j = s_j + blockIdx.x;
    int r = threadIdx.y;
    int c = threadIdx.x;
    int loc[div_block][div_block];

    if (i >= k)
        i++;
    if (j >= k)
        j++;

#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            i_k_sm[r + rr * TILE][c + cc * TILE] = _ref_blk(i, k, r + rr * TILE, c + cc * TILE);
        }
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_j_sm[r + rr * TILE][c + cc * TILE] = _ref_blk(k, j, r + rr * TILE, c + cc * TILE);
        }
    }
    __syncthreads();
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            loc[rr][cc] = _ref_blk(i, j, r + rr * TILE, c + cc * TILE);
        }
    }

#pragma unroll
    for (int b = 0; b < block_size; b++) {
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                loc[rr][cc] = min(loc[rr][cc], i_k_sm[r + rr * TILE][b] + k_j_sm[b][c + cc * TILE]);
            }
        }
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            _ref_blk(i, j, r + rr * TILE, c + cc * TILE) = loc[rr][cc];
        }
    }
}
__global__ void init_blk_dist(int *blk_dist, int pitch) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    blk_dist[r * pitch + c] = (r != c) * INF;
}
__global__ void build_blk_dist(int *edge, int E, int *blk_dist, int pitch) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < E) {
        int src = *(edge + idx * 3);
        int dst = *(edge + idx * 3 + 1);
        int w = *(edge + idx * 3 + 2);
        blk_dist[src * pitch + dst] = w;
    }
}